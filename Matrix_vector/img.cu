#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>

using namespace std;
using namespace cv;

__global__
void GScale(Mat* img, int iRow, int iCol){

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	if (col < iCol && row < iRow){

		// int rgbOffset = (col*iCol + row) * 3;
		//
		// img.at<Vec3f>(row, col)[0] *= 0.07f;		//BLUE
		// img.at<Vec3f>(row, col)[1] = 0.71f;		//GREEN
		// img.at<Vec3f>(row, col)[2] = 0.21f;		//RED
	}
}

__host__
int main(){

	Mat* d_Image;

	Mat image;
	image = imread("tux.bmp", CV_LOAD_IMAGE_COLOR);

	namedWindow( "Display window", WINDOW_AUTOSIZE );
	imshow( "Display window", image );

	// hipMalloc( (void**) &d_Image, sizeof(Mat));
	// hipMemcpy(d_Image, image, sizeof(Mat), hipMemcpyHostToDevice);

	cout<<image.at<Vec3b>(0, 0)<<endl;
	// for(int y = 0; y < image.rows; y++) {
	// 	for(int x = 0; x < image.cols; x++) {
	// 		image.at<Vec3b>(y, x)[0] *= 0.3;		//BLUE
	// 		image.at<Vec3b>(y, x)[1] *= 0.3;		//GREEN
	// 		image.at<Vec3b>(y, x)[2] *= 0.3;		//RED
	// 	}
	// }
	// GScale<<<1, 96>>>(image, image.rows, image.cols);

	// hipMemcpy(image, d_Image, sizeof(Mat), hipMemcpyDeviceToHost);

	namedWindow( "Display window GrayScale", WINDOW_AUTOSIZE );
	imshow( "Display window GrayScale", image );
	waitKey(0);

	return 0;
}
